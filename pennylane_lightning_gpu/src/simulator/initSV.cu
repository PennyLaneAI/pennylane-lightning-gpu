#include "hip/hip_runtime.h"
#include "cuda_helpers.hpp"
#include <hip/hip_complex.h>

namespace Pennylane {
template <class GPUDataT, class index_type>
__global__ void setStateVectorkernel(GPUDataT *sv, index_type num_indices,
                                     GPUDataT *value, index_type *indices) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_indices) {
        sv[indices[i]] = value[i];
    }
}

template <class GPUDataT, class index_type>
void setStateVector_CUDA(GPUDataT *sv, index_type &num_indices, GPUDataT *value,
                         index_type *indices, size_t thread_per_block,
                         hipStream_t stream_id) {
    auto dv = std::div(num_indices, thread_per_block);
    const size_t num_blocks = dv.quot + (dv.rem == 0 ? 0 : 1);
    dim3 blockSize(thread_per_block, 1, 1);
    dim3 gridSize(num_blocks, 1);

    setStateVectorkernel<GPUDataT, index_type>
        <<<gridSize, blockSize, 0, stream_id>>>(sv, num_indices, value,
                                                indices);
    PL_CUDA_IS_SUCCESS(hipGetLastError());
}

template <class GPUDataT>
void setBasisState_CUDA(GPUDataT *sv, GPUDataT &value, const size_t index,
                        bool async, hipStream_t stream_id) {
    if (!async) {
        PL_CUDA_IS_SUCCESS(hipMemcpy(&sv[index], &value, sizeof(GPUDataT),
                                      hipMemcpyHostToDevice));
    } else {
        PL_CUDA_IS_SUCCESS(hipMemcpyAsync(&sv[index], &value, sizeof(GPUDataT),
                                           hipMemcpyHostToDevice, stream_id));
    }
}

//} // namespace
//
// namespace Pennylane {
void setStateVector_CUDA(hipComplex *sv, int &num_indices, hipComplex *value,
                         int *indices, size_t thread_per_block,
                         hipStream_t stream_id);
void setStateVector_CUDA(hipDoubleComplex *sv, long &num_indices,
                         hipDoubleComplex *value, long *indices,
                         size_t thread_per_block, hipStream_t stream_id);

void setBasisState_CUDA(hipComplex *sv, hipComplex &value, const size_t index,
                        bool async, hipStream_t stream_id);
void setBasisState_CUDA(hipDoubleComplex *sv, hipDoubleComplex &value,
                        const size_t index, bool async, hipStream_t stream_id);

} // namespace Pennylane
#include "hip/hip_runtime.h"
// Copyright 2022 Xanadu Quantum Technologies Inc.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//     http://www.apache.org/licenses/LICENSE-2.0
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
/**
 * @file initSV.cu
 */
#include "cuda_helpers.hpp"
#include <hip/hip_complex.h>

namespace Pennylane {

void setStateVector_CUDA(hipComplex *sv, int &num_indices, hipComplex *value,
                         int *indices, size_t thread_per_block,
                         hipStream_t stream_id);
void setStateVector_CUDA(hipDoubleComplex *sv, long &num_indices,
                         hipDoubleComplex *value, long *indices,
                         size_t thread_per_block, hipStream_t stream_id);

void setBasisState_CUDA(hipComplex *sv, hipComplex &value, const size_t index,
                        bool async, hipStream_t stream_id);
void setBasisState_CUDA(hipDoubleComplex *sv, hipDoubleComplex &value,
                        const size_t index, bool async, hipStream_t stream_id);

template <class GPUDataT, class index_type>
__global__ void setStateVectorkernel(GPUDataT *sv, index_type num_indices,
                                     GPUDataT *value, index_type *indices) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_indices) {
        sv[indices[i]] = value[i];
    }
}

template <class GPUDataT, class index_type>
void setStateVector_CUDA_call(GPUDataT *sv, index_type &num_indices,
                              GPUDataT *value, index_type *indices,
                              size_t thread_per_block, hipStream_t stream_id) {
    auto dv = std::div(num_indices, thread_per_block);
    const size_t num_blocks = dv.quot + (dv.rem == 0 ? 0 : 1);
    dim3 blockSize(thread_per_block, 1, 1);
    dim3 gridSize(num_blocks, 1);

    setStateVectorkernel<GPUDataT, index_type>
        <<<gridSize, blockSize, 0, stream_id>>>(sv, num_indices, value,
                                                indices);
    PL_CUDA_IS_SUCCESS(hipGetLastError());
}

template <class GPUDataT>
void setBasisState_CUDA_call(GPUDataT *sv, GPUDataT &value, const size_t index,
                             bool async, hipStream_t stream_id) {
    if (!async) {
        PL_CUDA_IS_SUCCESS(hipMemcpy(&sv[index], &value, sizeof(GPUDataT),
                                      hipMemcpyHostToDevice));
    } else {
        PL_CUDA_IS_SUCCESS(hipMemcpyAsync(&sv[index], &value, sizeof(GPUDataT),
                                           hipMemcpyHostToDevice, stream_id));
    }
}

void setStateVector_CUDA(hipComplex *sv, int &num_indices, hipComplex *value,
                         int *indices, size_t thread_per_block,
                         hipStream_t stream_id) {
    setStateVector_CUDA_call(sv, num_indices, value, indices, thread_per_block,
                             stream_id);
}
void setStateVector_CUDA(hipDoubleComplex *sv, long &num_indices,
                         hipDoubleComplex *value, long *indices,
                         size_t thread_per_block, hipStream_t stream_id) {
    setStateVector_CUDA_call(sv, num_indices, value, indices, thread_per_block,
                             stream_id);
}

void setBasisState_CUDA(hipComplex *sv, hipComplex &value, const size_t index,
                        bool async, hipStream_t stream_id) {
    setBasisState_CUDA_call(sv, value, index, async, stream_id);
}
void setBasisState_CUDA(hipDoubleComplex *sv, hipDoubleComplex &value,
                        const size_t index, bool async,
                        hipStream_t stream_id) {
    setBasisState_CUDA_call(sv, value, index, async, stream_id);
}

} // namespace Pennylane